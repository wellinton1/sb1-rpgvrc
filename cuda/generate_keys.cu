
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C" __global__ void generateKeys(unsigned char* keys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize CUDA random number generator with better entropy
    hiprandState state;
    hiprand_init(clock64() + idx, threadIdx.x, blockIdx.x, &state);
    
    // Generate 32 bytes for private key with improved entropy
    unsigned char key[32];
    for(int i = 0; i < 32; i++) {
        // Mix multiple sources of entropy
        unsigned int r1 = hiprand(&state);
        unsigned int r2 = hiprand(&state);
        unsigned int r3 = clock64() ^ r1;
        key[i] = (r1 ^ r2 ^ r3) % 256;
    }
    
    // Write to global memory
    for(int i = 0; i < 32; i++) {
        keys[idx * 32 + i] = key[i];
    }
    
    __syncthreads();
}